#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

const int W = 40;
const int H = 12;

__global__ void Draw(char *frame) {
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (y < H and x < W) {
		char c;
		if (x == W-1) {
			c = y == H-1 ? '\0' : '\n';
		} else if (y == 0 or y == H-1 or x == 0 or x == W-2) {
			c = ':';
		} else if (y > 4 and y < H-1) {
			if ((x == 33 and y == H-2) or (x < 22 and x > H-2*y+15)) {
				c = '#';
			}
			else if (x == 33) {
				c = '|';
			}
			else if (y == 5 and x == 32) {
				c = '<';
			}
			else {
				c = ' ';
			}
		} else {
			c = ' ';
		}
		frame[y*W+x] = c;
	}
}

int main(int argc, char **argv)
{
	MemoryBuffer<char> frame(W*H);
	auto frame_smem = frame.CreateSync(W*H);
	CHECK;

	Draw<<<dim3((W-1)/16+1,(H-1)/12+1), dim3(16,12)>>>(frame_smem.get_gpu_wo());
	CHECK;

	puts(frame_smem.get_cpu_ro());
	CHECK;
	return 0;
}
